#include "to_lavc.h"
#include "from_lavc.h"

#include <vector>
#include <stdlib.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <chrono>
#include "from_lavc.h"
#include <string>

using std::chrono::milliseconds;
using namespace std::string_literals;

int main(int argc, char *argv[]){
    if (argc != 6){
        printf("bad input\n <width> <height> <in_name> <in_codec> <out_codec>\n");
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    codec_t UG_codec = get_codec_from_file_extension(argv[4]);
    AVPixelFormat AV_codec = av_get_pix_fmt(argv[5]);
    assert(AV_codec != AV_PIX_FMT_NONE && UG_codec != VIDEO_CODEC_NONE);

    std::ifstream fin(argv[3], std::ifstream::binary);
    std::ofstream fout1("AVtest_"s + argv[5] + ".rgba", std::ofstream::binary);
    std::ofstream reference("AVreference_"s + argv[5] + ".rgba", std::ofstream::binary);
    assert (width && height && fin && fout1 && reference);

    size_t in_size = vc_get_datalen(width, height, RGBA);
    std::vector<unsigned char> fin_data(in_size);
    fin.read(reinterpret_cast<char *>(fin_data.data()), in_size);


    //RGBA -> RG48 because it has conversion to every UG format
    std::vector<unsigned char> rg48vec(vc_get_datalen(width, height, RG48));
    auto d = get_decoder_from_to(RGBA, RG48);
    for (int y = 0; y < height; ++y){
        d(rg48vec.data() + y * vc_get_linesize(width, RG48),
               fin_data.data()+ y * vc_get_linesize(width, RGBA),
               vc_get_linesize(width, RG48), 0, 8, 16);
    }

    //rg48 -> ug codec
    auto decode = get_decoder_from_to(RG48, UG_codec);
    if (decode == NULL){
        std::cout << "cannot find RGBA -> UG format";
        return 1;
    }
    std::vector<unsigned char> UG_converted(vc_get_datalen(width, height, UG_codec));
    for (int y = 0; y < height; ++y){
        decode(UG_converted.data() + y * vc_get_linesize(width, UG_codec),
               rg48vec.data() + y * vc_get_linesize(width, RG48),
               vc_get_linesize(width, UG_codec), 0, 8, 16);
    }

    std::cout << AV_codec << '\n';
    std::cout.flush();

    //convert UG -> AV
    //-------------------------------------------gpu version
    AVFrame *frame1 = nullptr;
    char *dst_cpu1 = nullptr;
    if (to_lavc_init(AV_codec, UG_codec, width, height, &frame1)){
        convert_to_lavc(UG_codec, frame1, reinterpret_cast<char *>(UG_converted.data()));
        if (from_lavc_init(frame1, RGBA, &dst_cpu1))
            convert_from_lavc(frame1, dst_cpu1, RGBA);
    } else {
        std::cout << "non-existing gpu implementation\n";
    }

    //-------------------------------------------cpu version
    struct to_lavc_vid_conv *conv_to_av = to_lavc_vid_conv_init(UG_codec, width, height, AV_codec, 1);
    char *dst_cpu2 = nullptr;
    if (conv_to_av){
        AVFrame *frame2 = to_lavc_vid_conv(conv_to_av, (char *) UG_converted.data());
        if (from_lavc_init(frame2, RGBA, &dst_cpu2))
            convert_from_lavc(frame2, dst_cpu2, RGBA);
    } else {
        std::cout << "non-existing cpu implementation\n";
    }

    //--------------------------------

    fout1.write(dst_cpu1, vc_get_datalen(width, height, RGBA));
    reference.write(dst_cpu2, vc_get_datalen(width, height, RGBA));
    std::cout << hipGetErrorString(hipGetLastError());
}
