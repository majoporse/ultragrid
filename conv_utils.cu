#include "conv_utils.h"
#include <iostream>

AVF_GPU_wrapper::AVF_GPU_wrapper(){
    frame.data[0] = nullptr;
    frame.data[1] = nullptr;
    frame.data[2] = nullptr;
    frame.data[3] = nullptr;
}

void AVF_GPU_wrapper::alloc(const AVFrame* new_frame) {
    frame = *new_frame;
    if (new_frame->format == AV_PIX_FMT_YUV420P ||
        new_frame->format == AV_PIX_FMT_YUV420P10LE ||
        new_frame->format == AV_PIX_FMT_YUV420P12LE ||
        new_frame->format == AV_PIX_FMT_YUV420P16LE ||
        new_frame->format == AV_PIX_FMT_NV12 ||
        new_frame->format == AV_PIX_FMT_YUVJ420P ||
        new_frame->format == AV_PIX_FMT_P010LE)
        q = 2;

    hipMalloc(&(frame.data[0]), frame.linesize[0] * frame.height);
    hipMalloc(&(frame.data[1]), frame.linesize[1] * frame.height / q);
    hipMalloc(&(frame.data[2]), frame.linesize[2] * frame.height / q);
    hipMalloc(&(frame.data[3]), frame.linesize[3] * frame.height);
}

void AVF_GPU_wrapper::copy_to_device(const AVFrame *new_frame){
    hipMemcpy(frame.data[0], new_frame->data[0], frame.linesize[0] * frame.height, hipMemcpyHostToDevice);
    if (frame.linesize[1]) hipMemcpy(frame.data[1], new_frame->data[1], frame.linesize[1] * frame.height / q, hipMemcpyHostToDevice);
    if (frame.linesize[2]) hipMemcpy(frame.data[2], new_frame->data[2], frame.linesize[2] * frame.height / q, hipMemcpyHostToDevice);
    if (frame.linesize[3]) hipMemcpy(frame.data[3], new_frame->data[3], frame.linesize[3] * frame.height, hipMemcpyHostToDevice);
}

void AVF_GPU_wrapper::copy_to_host(const AVFrame *new_frame){
                           hipMemcpy( new_frame->data[0],frame.data[0], frame.linesize[0] * frame.height, hipMemcpyDeviceToHost);
    if (frame.linesize[1]) hipMemcpy( new_frame->data[1],frame.data[1], frame.linesize[1] * frame.height / q, hipMemcpyDeviceToHost);
    if (frame.linesize[2]) hipMemcpy( new_frame->data[2],frame.data[2], frame.linesize[2] * frame.height / q, hipMemcpyDeviceToHost);
    if (frame.linesize[3]) hipMemcpy( new_frame->data[3],frame.data[3], frame.linesize[3] * frame.height, hipMemcpyDeviceToHost);
}

void AVF_GPU_wrapper::free_from_device(){
    hipFree(frame.data[0]);
    hipFree(frame.data[1]);
    hipFree(frame.data[2]);
    hipFree(frame.data[3]);

    frame.data[0] = nullptr;
    frame.data[1] = nullptr;
    frame.data[2] = nullptr;
    frame.data[3] = nullptr;
    q = 1;
}